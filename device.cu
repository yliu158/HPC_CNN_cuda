#include "hip/hip_runtime.h"
#include "main.h"

__global__ void full_forward(int* weight, int* input, int* output) {

}

template <typename IN_DIMS, size_t N_NEURONS>
void
FullyConnectedLayer<IN_DIMS, N_NEURONS>::device_forward(const Input &input, const Array<Input, N_NEURONS> &weight, const Array<double, N_NEURONS> &bias, const Array<double, N_NEURONS> &dropped, Output &output) {
  printf("test of cuda forward function.");
  int *w, *i, *o;
  w = (int*)malloc(sizeof(int)); 
  i = (int*)malloc(sizeof(int));
  o = (int*)malloc(sizeof(int));
  int *d_w, *d_i, *d_o;
  hipMalloc((int**)&d_w, sizeof(int));
  hipMalloc((int**)&d_i, sizeof(int));
  hipMalloc((int**)&d_o, sizeof(int));
  hipMemcpy(d_w, w, sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_i, i, sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_o, o, sizeof(int),hipMemcpyHostToDevice);

  free(w);
  free(i);
  free(o);
}
// __global__ void conv_forward() {
//
// }
//
// __global__ void padding(float* origin, float* padded) {
//   // int blockId = blockIdx.x + blockDim.x*blockIdx.y;
//   // int threadId = threadIdx.x + threadDim.x*threadIdx.y + blockId*threadDim.x*threadDim.y;
//   int threadId = threadIdx.x + threadDim.x * threadidx.y;
//   if (threadIdx.x < 2 || threadIdx.x > 29 || threadIdx.y < 2 || threadIdx.y > 29) {
//     padded[threadId] = 0.0;
//     return;
//   }
//   int ori_id = (threadIdx.x-2) + (threadDim.x-2)*(threadIdx.y-2);
//   padded[threadId] = origin[ori_id];
// }
//
// template <typename IN_DIMS, size_t N_FILTERS>
// void
// conv_forward_device() {
//     // dim3 grids(2,2,1);
//     // dim3 blocks(16, 16, 1);
//     dim3 grids(1,1,1);
//     dim3 blocks(32, 32, 1);
//     float *in, *out;
//     float *d_in, *d_out;
//     in = (float*)malloc(sizeof(float)*28*28);
//     out = (float*)malloc(sizeof(float)*32*32);
//     hipMalloc((float**)&d_in, sizeof(float)*28*28);
//     hipMalloc((float**)&d_out, sizeof(float)*32*32);
//     hipMemcpy(d_in, in, sizeof(float)*28*28, hipMemcpyHostToDevice);
//     hipMemcpy(d_out, out, sizeof(float)*32*32, hipMemcpyHostToDevice);
//
//     padding<<<grids,blocks>>>(in, out);
//
//     __syncthreads();
//
//     hipMemcpy(d_out, out, sizeof(float)*32*32, hipMemcpyDeviceToHost);
//     free(in); free(out);
//     hipFree(d_in); hipFree(d_out);
//     // __syncthreads();
// }
