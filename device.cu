#include "hip/hip_runtime.h"
#include "main.h"

__global__ void pool_forward(double* in, double* out) {
  int t_id = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.y*blockDim.x;
  int o_id = threadIdx.x/2 + threadIdx.y/2*(blockDim.x/2) + threadIdx.z*(blockDim.y/2)*(blockDim.x/2);
  if (in[t_id] > out[o_id]) {
    out[o_id] = 8;
  } else {
    out[o_id] = 0.1111;
  }
}

void pool_device_forward(double* in, double* out) {
  dim3 block_size(28*28*32);
  double *d_in, *d_out;
  hipMalloc((double**)&d_in, sizeof(double)*28*28*32);
  hipMalloc((double**)&d_out, sizeof(double)*14*14*32);
  hipMemcpy(d_in, in, sizeof(double)*28*28*32, hipMemcpyHostToDevice);
  pool_forward<<<1, block_size>>>(d_in, d_out);
  hipMemcpy(out, d_out, sizeof(double)*14*14*32, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

// __global__ void conv_forward(double* weight, double* input, double* output) {
//
// }
//
// void conv_device_forward(double * w, double * i, double * o) {
//
// }



// __global__ void full_forward(double* weight, double* input, double* output) {
//   int threadId = threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;
//   int blockId = blockIdx.x;
//   output[blockId] += weight[threadId+blockId]*input[threadId];
// }
//
// void full_device_forward(double * w, double * i, double * o) {
//   printf("test of cuda forward function.\n");
//   double *d_w, *d_i, *d_o;
//   hipMalloc((double**)&d_w, sizeof(double)*7*7*64*1024);
//   hipMalloc((double**)&d_i, sizeof(double)*7*7*64);
//   hipMalloc((double**)&d_o, sizeof(double)*1024);
//   hipMemcpy(d_w, w, sizeof(double)*7*7*64*1024,hipMemcpyHostToDevice);
//   hipMemcpy(d_i, i, sizeof(double)*7*7*64,hipMemcpyHostToDevice);
//   hipMemcpy(d_o, o, sizeof(double)*1024,hipMemcpyHostToDevice);
//   dim3 grid_size(1024,1,1);
//   dim3 block_size(7,7,64);
//   full_forward<<<grid_size, block_size>>>(d_w, d_i, d_o);
//   hipMemcpy(o, d_o, sizeof(double)*1024,hipMemcpyDeviceToHost);
//   hipFree(d_w);
//   hipFree(d_i);
//   hipFree(d_o);
// }
// __global__ void conv_forward() {
//
// }
//
// __global__ void padding(double* origin, double* padded) {
//   // double blockId = blockIdx.x + blockDim.x*blockIdx.y;
//   // double threadId = threadIdx.x + threadDim.x*threadIdx.y + blockId*threadDim.x*threadDim.y;
//   double threadId = threadIdx.x + threadDim.x * threadidx.y;
//   if (threadIdx.x < 2 || threadIdx.x > 29 || threadIdx.y < 2 || threadIdx.y > 29) {
//     padded[threadId] = 0.0;
//     return;
//   }
//   double ori_id = (threadIdx.x-2) + (threadDim.x-2)*(threadIdx.y-2);
//   padded[threadId] = origin[ori_id];
// }
//
// template <typename IN_DIMS, size_t N_FILTERS>
// void
// conv_forward_device() {
//     // dim3 grids(2,2,1);
//     // dim3 blocks(7*7*64*1024, 7*7*64*1024, 1);
//     dim3 grids(1,1,1);
//     dim3 blocks(32, 32, 1);
//     double *in, *out;
//     double *d_in, *d_out;
//     in = (double*)malloc(sizeof(double)*28*28);
//     out = (double*)malloc(sizeof(double)*32*32);
//     hipMalloc((double**)&d_in, sizeof(double)*28*28);
//     hipMalloc((double**)&d_out, sizeof(double)*32*32);
//     hipMemcpy(d_in, in, sizeof(double)*28*28, hipMemcpyHostToDevice);
//     hipMemcpy(d_out, out, sizeof(double)*32*32, hipMemcpyHostToDevice);
//
//     padding<<<grids,blocks>>>(in, out);
//
//     __syncthreads();
//
//     hipMemcpy(d_out, out, sizeof(double)*32*32, hipMemcpyDeviceToHost);
//     free(in); free(out);
//     hipFree(d_in); hipFree(d_out);
//     // __syncthreads();
// }
