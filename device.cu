#include "hip/hip_runtime.h"
#include "main.h"

// __global__ void pool_forward(double* in, double* out) {
//   int out_id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y;
//   int in_id = threadIdx.x*2 + (threadIdx.y*2)*(blockDim.x*2) + blockIdx.x*(blockDim.x*blockDim.y)*4;
//   out[out_id] = in[in_id];
//   for (int i = 0; i < 2; ++i) {
//     for (int j = 0; j < 2; ++j) {
//       if (out[out_id] < in[in_id+i+j*blockDim.x*2]) {
//         out[out_id] = in[in_id+i+j*blockDim.x*2];
//       }
//     }
//   }
// }

// void pool_forward_device_first(double* in, double* out) {
//   dim3 block_size(14,14,1);
//   dim3 grid_size(32,1,1);
//   double *d_in, *d_out;
//   hipMalloc((double**)&d_in, sizeof(double)*28*28*32);
//   hipMalloc((double**)&d_out, sizeof(double)*14*14*32);
//   hipMemcpy(d_in, in, sizeof(double)*28*28*32, hipMemcpyHostToDevice);
//
//   pool_forward<<<grid_size, block_size>>>(d_in, d_out);
//
//   hipMemcpy(out, d_out, sizeof(double)*14*14*32, hipMemcpyDeviceToHost);
//   hipFree(d_in);
//   hipFree(d_out);
// }
//
// void pool_forward_device_second(double* in, double* out) {
//   dim3 block_size(7,7,1);
//   dim3 grid_size(64,1,1);
//   double *d_in, *d_out;
//   hipMalloc((double**)&d_in, sizeof(double)*14*14*64);
//   hipMalloc((double**)&d_out, sizeof(double)*7*7*64);
//   hipMemcpy(d_in, in, sizeof(double)*14*14*64, hipMemcpyHostToDevice);
//
//   pool_forward<<<grid_size, block_size>>>(d_in, d_out);
//
//   hipMemcpy(out, d_out, sizeof(double)*7*7*64, hipMemcpyDeviceToHost);
//   hipFree(d_in);
//   hipFree(d_out);
// }

__global__ void pool_forward(double *in, double *out, size_t size_out) {
  int o_id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y;
  int i_id = threadIdx.x*2 + threadIdx.y*2*blockDim.x*2 + blockIdx.x*blockDim.x*2*blockDim.y*2;

  for (size_t i = 0; i < 2; i++) {
    for (size_t j = 0; j < 2; j++) {
      if (out[o_id] < in[i_id+i*2*size_out+j]) {
        out[o_id] = in[i_id+i*2*size_out+j];
      }
    }
  }
  // printf("Hello o_id:\n", threadIdx.x);
}

void pool_forward_device(double* in, double* out, size_t size_out, size_t img_d) {
  double *d_in, *d_out;
  hipMalloc((double**)&d_in, sizeof(double)*size_out*2*size_out*2*img_d);
  hipMalloc((double**)&d_out, sizeof(double)*size_out*size_out*img_d);
  hipMemcpy(d_in, in, sizeof(double)*size_out*2*size_out*2*img_d, hipMemcpyHostToDevice);

  dim3 block_size(size_out, size_out, 1);
  dim3 grid_size(img_d, 1, 1);
  pool_forward<<<grid_size, block_size>>>(d_in, d_out, size_out);

  hipMemcpy(out, d_out, sizeof(double)*size_out*size_out*img_d, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}



// __global__ void conv_forward(double* in, double* filter, double* bias, double* out) {
//   int t_id = threadIdx.x + threadIdx.y*blockDim.x + blockDim.x*blockDim.y*blockIdx.x;
//   int i_id = threadIdx.x+2 + threadIdx.y*(blockDim.x+4) + (blockDim.x+4)*(blockDim.y+4)*blockIdx.x;
//   double res = 0;
//   for (int i = -2; i <= 2; ++i) {
//     for (int j = -2; j <= 2; ++j) {
//       res += in[i_id+i*32+j]*filter[blockIdx.x*25+i*5+j];
//     }
//   }
//   out[t_id] = res + bias[blockIdx.x];
//   // printf("tid: %d\n", t_id);
// }
//
// void conv_forward_device_first(double* in, double* filter, double* bias, double* out) {
//   double *d_i, *d_f, *d_b, *d_o;
//   hipMalloc((double**)&d_i, sizeof(double)*32*32*1);
//   hipMalloc((double**)&d_f, sizeof(double)*5*5*32);
//   hipMalloc((double**)&d_b, sizeof(double)*32);
//   hipMalloc((double**)&d_o, sizeof(double)*28*28*32);
//   hipMemcpy(d_i, in, sizeof(double)*32*32*1, hipMemcpyHostToDevice);
//   hipMemcpy(d_f, filter, sizeof(double)*5*5*32, hipMemcpyHostToDevice);
//   hipMemcpy(d_b, bias, sizeof(double)*32, hipMemcpyHostToDevice);
//
//   dim3 block_size(28,28,1);
//   dim3 grid_size(32,1,1);
//   conv_forward<<<grid_size, block_size>>>(d_i, d_f, d_b, d_o);
//
//   hipMemcpy(out, d_o, sizeof(double)*28*28*32, hipMemcpyDeviceToHost);
//   hipFree(d_i);
//   hipFree(d_f);
//   hipFree(d_b);
//   hipFree(d_o);
// }

__global__ void conv_forward_all(double* in, double* filter, double* bias, double* out) {
  // gridDim.x:1  blockDim.x:3  blockDim.y:3  gridDim.y:32
  int x_out = threadIdx.x;
  int y_out = threadIdx.y*blockDim.x;
  int z_out = blockIdx.x*blockDim.x*blockDim.y;
  int w_out = blockIdx.y*gridDim.x*blockDim.x*blockDim.y;
  int o_id = x_out + y_out + z_out + w_out;
  int x_in = threadIdx.x+2;// 3
  int y_in = (threadIdx.y+2)*(blockDim.x+4); //21
  int z_in = blockIdx.x*(blockDim.x+4)*(blockDim.y+4);
  int i_id = x_in + y_in + z_in;
  for (int i = -2; i <= 2; ++i) {
    for (int j = -2; j <= 2; ++j) {
      out[o_id] += filter[blockIdx.y*25*gridDim.x+(i+2)*5+j+2] * in[i_id+i*(blockDim.x+4)+j+2];
    }
  }
  // out[o_id] += bias[blockIdx.y];
  // printf("%lf\n", out[o_id]);
}

void conv_forward_device(double* in, double* filter, double* bias, double* out, size_t size, size_t img_d, size_t fil_d) {
  double *d_i, *d_f, *d_b, *d_o;
  hipMalloc((double**)&d_i, sizeof(double)*(size+4)*(size+4)*img_d);
  hipMalloc((double**)&d_f, sizeof(double)*5*5*img_d*fil_d);
  hipMalloc((double**)&d_b, sizeof(double)*fil_d);
  hipMalloc((double**)&d_o, sizeof(double)*size*size*fil_d);
  hipMemcpy(d_i, in, sizeof(double)*(size+4)*(size+4)*img_d, hipMemcpyHostToDevice);
  hipMemcpy(d_f, filter, sizeof(double)*5*5*img_d*fil_d, hipMemcpyHostToDevice);
  hipMemcpy(d_b, bias, sizeof(double)*fil_d, hipMemcpyHostToDevice);
  dim3 block_size(size,size,1);
  dim3 grid_size(img_d,fil_d,1);
  conv_forward_all<<<grid_size, block_size>>>(d_i, d_f, d_b, d_o);
  hipMemcpy(out, d_o, sizeof(double)*size*size*fil_d, hipMemcpyDeviceToHost);
  hipFree(d_i);
  hipFree(d_f);
  hipFree(d_b);
  hipFree(d_o);
}


// __global__ void conv_forward_test(double* in, double* filter, double* bias, double* out) {
//   int t_id = threadIdx.x + threadIdx.y*blockDim.x + blockDim.x*blockDim.y*blockIdx.x;
//   int i_id = threadIdx.x+1 + threadIdx.y*(blockDim.x+2) + (blockDim.x+2)*(blockDim.y+2)*blockIdx.x;
//   double res = 0;
//   for (int i = -1; i <= 1; ++i) {
//     for (int j = -1; j <= 1; ++j) {
//       res += in[i_id+i*5+j]*filter[blockIdx.x*9+i*3+j];
//     }
//   }
//   out[t_id] = res + bias[blockIdx.x];
//   // printf("tid: %d\n", t_id);
// }
//
// void conv_forward_device_test(double* in, double* filter, double* bias, double* out) {
//   double *d_i, *d_f, *d_b, *d_o;
//   hipMalloc((double**)&d_i, sizeof(double)*5*5*1);
//   hipMalloc((double**)&d_f, sizeof(double)*3*3*32);
//   hipMalloc((double**)&d_b, sizeof(double)*32);
//   hipMalloc((double**)&d_o, sizeof(double)*3*3*32);
//   hipMemcpy(d_i, in, sizeof(double)*5*5*1, hipMemcpyHostToDevice);
//   hipMemcpy(d_f, filter, sizeof(double)*3*3*32, hipMemcpyHostToDevice);
//   hipMemcpy(d_b, bias, sizeof(double)*32, hipMemcpyHostToDevice);
//
//   dim3 block_size(3,3,1);
//   dim3 grid_size(32,1,1);
//   conv_forward_test<<<grid_size, block_size>>>(d_i, d_f, d_b, d_o);
//
//   hipMemcpy(out, d_o, sizeof(double)*3*3*32, hipMemcpyDeviceToHost);
//   hipFree(d_i);
//   hipFree(d_f);
//   hipFree(d_b);
//   hipFree(d_o);
// }


// __global__ void add(int *x, int *y, int *z) {
//   z[threadIdx.x] = y[threadIdx.x] + x[threadIdx.x];
//   printf("Hello %d\n", threadIdx.x);
// }
//
// void test_device (int* x, int* y, int* z) {
//   int* d_x, *d_y, *d_z;
//   hipMalloc((int**)&d_x, sizeof(int)*16);
//   hipMalloc((int**)&d_y, sizeof(int)*16);
//   hipMalloc((int**)&d_z, sizeof(int)*16);
//   hipMemcpy(d_x, x, sizeof(int)*16, hipMemcpyHostToDevice);
//   hipMemcpy(d_y, y, sizeof(int)*16, hipMemcpyHostToDevice);
//
//   add<<<1,16>>> (d_x, d_y, d_z);
//
//   hipMemcpy(z, d_z, sizeof(int)*16, hipMemcpyDeviceToHost);
//   hipFree(d_x);
//   hipFree(d_y);
//   hipFree(d_z);
// }
