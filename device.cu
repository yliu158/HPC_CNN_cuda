#include "hip/hip_runtime.h"
#include "main.h"

__global__ void pool_forward(double *in, double *out, size_t size_out) {
  int o_id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y;
  int i_id = threadIdx.x*2 + threadIdx.y*2*blockDim.x*2 + blockIdx.x*blockDim.x*2*blockDim.y*2;

  for (size_t i = 0; i < 2; i++) {
    for (size_t j = 0; j < 2; j++) {
      if (out[o_id] < in[i_id+i*2*size_out+j]) {
        out[o_id] = in[i_id+i*2*size_out+j];
      }
    }
  }
}

void pool_forward_device(double* in, double* out, size_t size_out, size_t img_d) {
  double *d_in, *d_out;
  hipMalloc((double**)&d_in, sizeof(double)*size_out*2*size_out*2*img_d);
  hipMalloc((double**)&d_out, sizeof(double)*size_out*size_out*img_d);
  hipMemcpy(d_in, in, sizeof(double)*size_out*2*size_out*2*img_d, hipMemcpyHostToDevice);

  dim3 block_size(size_out, size_out, 1);
  dim3 grid_size(img_d, 1, 1);
  pool_forward<<<grid_size, block_size>>>(d_in, d_out, size_out);

  hipMemcpy(out, d_out, sizeof(double)*size_out*size_out*img_d, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}


__global__ void conv_forward(double* in, double* filter, double* bias, double* out) {
  int i_id = (threadIdx.x+2)+(threadIdx.y+2)*(blockDim.x+4)+blockIdx.x*(blockDim.x+4)*(blockDim.y+4);
  int o_id = threadIdx.x+threadIdx.y*blockDim.x+blockIdx.y*blockDim.x*blockDim.y;
  int f_id = 12+blockIdx.x*25+blockIdx.y*25*gridDim.x;
  for (int i = -2; i < 3; i++) {
    for (int j = -2; j < 3; j++) {
      out[o_id] += in[i_id+i*(blockDim.x+4)+j]*filter[f_id+i*5+j];
    }
  }
  out[o_id] += bias[blockIdx.y];
  if (out[o_id] < 0) out[o_id] = 0.0;
}


void conv_forward_device(double* in, double* filter, double* bias, double* out, size_t size, size_t img_d, size_t fil_d) {
  double *d_i, *d_f, *d_b, *d_o;
  hipMalloc((double**)&d_i, sizeof(double)*(size+4)*(size+4)*img_d);
  hipMalloc((double**)&d_f, sizeof(double)*5*5*img_d*fil_d);
  hipMalloc((double**)&d_b, sizeof(double)*fil_d);
  hipMalloc((double**)&d_o, sizeof(double)*size*size*fil_d);
  hipMemcpy(d_i, in, sizeof(double)*(size+4)*(size+4)*img_d, hipMemcpyHostToDevice);
  hipMemcpy(d_f, filter, sizeof(double)*5*5*img_d*fil_d, hipMemcpyHostToDevice);
  hipMemcpy(d_b, bias, sizeof(double)*fil_d, hipMemcpyHostToDevice);
  dim3 block_size(size,size,1);
  dim3 grid_size(img_d,fil_d,1);
  conv_forward<<<grid_size, block_size>>>(d_i, d_f, d_b, d_o);
  hipMemcpy(out, d_o, sizeof(double)*size*size*fil_d, hipMemcpyDeviceToHost);
  hipFree(d_i);
  hipFree(d_f);
  hipFree(d_b);
  hipFree(d_o);
}

__global__ void full_forward_conv(double * in, double * out, double * weight) {
  int i_id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y;
  int o_id = blockIdx.y;
  int w_id = i_id + blockIdx.y*gridDim.x*blockDim.x*blockDim.y;
  out[o_id] += in[i_id]*weight[w_id];
  printf("%d\n", o_id);
}

__global__ void full_forward_bias_drop(double * out, double * bias, double * drop){
  out[threadIdx.x] += bias[threadIdx.x];
  if (out[threadIdx.x] < 0) out[threadIdx.x] = 0.0;
  out[threadIdx.x] *= drop[threadIdx.x];
}

void full_forward_device(double * in, double * out, double * weight, double* bias, double* drop, size_t size, size_t img_d, size_t n_nro) {
  double *d_in, *d_out, *d_weight, *d_bias, *d_drop;
  hipMalloc((double**)&d_in, sizeof(double)*size*size*img_d);
  hipMalloc((double**)&d_out, sizeof(double)*n_nro);
  hipMalloc((double**)&d_weight, sizeof(double)*size*size*img_d*n_nro);
  hipMalloc((double**)&d_bias, sizeof(double)*n_nro);
  hipMalloc((double**)&d_drop, sizeof(double)*n_nro);
  hipMemcpy(d_in, in, sizeof(double)*size*size*img_d, hipMemcpyHostToDevice);
  hipMemcpy(d_weight, weight, sizeof(double)*size*size*img_d*n_nro, hipMemcpyHostToDevice);
  hipMemcpy(d_bias, bias, sizeof(double)*n_nro, hipMemcpyHostToDevice);
  hipMemcpy(d_drop, drop, sizeof(double)*n_nro, hipMemcpyHostToDevice);

  dim3 block_size(size, size, 1);
  dim3 grid_size(img_d, n_nro, 1);
  full_forward_conv<<<grid_size,block_size>>>(d_in, d_out, d_weight);
  // full_forward_bias_drop<<<1,n_nro>>>(d_out, d_bias, d_drop);
  hipMemcpy(out, d_out, sizeof(double)*n_nro, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_weight);
  hipFree(d_bias);
  hipFree(d_drop);
}
