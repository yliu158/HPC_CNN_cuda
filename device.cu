#include "hip/hip_runtime.h"
#include "main.h"

__global__ void pool_forward(double *in, double *out, size_t size_out) {
  int o_id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y;
  int i_id = threadIdx.x*2 + threadIdx.y*2*blockDim.x*2 + blockIdx.x*blockDim.x*2*blockDim.y*2;

  for (size_t i = 0; i < 2; i++) {
    for (size_t j = 0; j < 2; j++) {
      if (out[o_id] < in[i_id+i*2*size_out+j]) {
        out[o_id] = in[i_id+i*2*size_out+j];
      }
    }
  }
}

void pool_forward_device(double* in, double* out, size_t size_out, size_t img_d) {
  double *d_in, *d_out;
  hipMalloc((double**)&d_in, sizeof(double)*size_out*2*size_out*2*img_d);
  hipMalloc((double**)&d_out, sizeof(double)*size_out*size_out*img_d);
  hipMemcpy(d_in, in, sizeof(double)*size_out*2*size_out*2*img_d, hipMemcpyHostToDevice);

  dim3 block_size(size_out, size_out, 1);
  dim3 grid_size(img_d, 1, 1);
  pool_forward<<<grid_size, block_size>>>(d_in, d_out, size_out);

  hipMemcpy(out, d_out, sizeof(double)*size_out*size_out*img_d, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}


__global__ void conv_forward(double* in, double* filter, double* bias, double* out) {
  int i_id = (threadIdx.x+2)+(threadIdx.y+2)*(blockDim.x+4)+blockIdx.x*(blockDim.x+4)*(blockDim.y+4);
  int o_id = threadIdx.x+threadIdx.y*blockDim.x+blockIdx.y*blockDim.x*blockDim.y;
  int f_id = 12+blockIdx.x*25+blockIdx.y*25*gridDim.x;
  for (int i = -2; i < 3; i++) {
    for (int j = -2; j < 3; j++) {
      out[o_id] += in[i_id+i*(blockDim.x+4)+j]*filter[f_id+i*5+j];
    }
  }
  out[o_id] += bias[blockIdx.y];
  if (out[o_id] < 0) out[o_id] = 0.0;
}


void conv_forward_device(double* in, double* filter, double* bias, double* out, size_t size, size_t img_d, size_t fil_d) {
  double *d_i, *d_f, *d_b, *d_o;
  hipMalloc((double**)&d_i, sizeof(double)*(size+4)*(size+4)*img_d);
  hipMalloc((double**)&d_f, sizeof(double)*5*5*img_d*fil_d);
  hipMalloc((double**)&d_b, sizeof(double)*fil_d);
  hipMalloc((double**)&d_o, sizeof(double)*size*size*fil_d);
  hipMemcpy(d_i, in, sizeof(double)*(size+4)*(size+4)*img_d, hipMemcpyHostToDevice);
  hipMemcpy(d_f, filter, sizeof(double)*5*5*img_d*fil_d, hipMemcpyHostToDevice);
  hipMemcpy(d_b, bias, sizeof(double)*fil_d, hipMemcpyHostToDevice);
  dim3 block_size(size,size,1);
  dim3 grid_size(img_d,fil_d,1);
  conv_forward<<<grid_size, block_size>>>(d_i, d_f, d_b, d_o);
  hipMemcpy(out, d_o, sizeof(double)*size*size*fil_d, hipMemcpyDeviceToHost);
  hipFree(d_i);
  hipFree(d_f);
  hipFree(d_b);
  hipFree(d_o);
}


__global__ void full_forward() {

}

void full_forward_device(double * w, double * i, double * o) {

}
