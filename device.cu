#include "hip/hip_runtime.h"
#include "main.h"

__global__ void pool_forward(double* in, double* out) {
  int out_id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y;
  int in_id = threadIdx.x*2 + (threadIdx.y*2)*(blockDim.x*2) + blockIdx.x*(blockDim.x*blockDim.y)*4;
  out[out_id] = in[in_id];
  for (int i = 0; i < 2; ++i) {
    for (int j = 0; j < 2; ++j) {
      if (out[out_id] < in[in_id+i+j*blockDim.x*2]) {
        out[out_id] = in[in_id+i+j*blockDim.x*2];
      }
    }
  }
}

void pool_forward_device_first(double* in, double* out) {
  dim3 block_size(14,14,1);
  dim3 grid_size(32,1,1);
  double *d_in, *d_out;
  hipMalloc((double**)&d_in, sizeof(double)*28*28*32);
  hipMalloc((double**)&d_out, sizeof(double)*14*14*32);
  hipMemcpy(d_in, in, sizeof(double)*28*28*32, hipMemcpyHostToDevice);

  pool_forward<<<grid_size, block_size>>>(d_in, d_out);

  hipMemcpy(out, d_out, sizeof(double)*14*14*32, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

void pool_forward_device_second(double* in, double* out) {
  dim3 block_size(7,7,1);
  dim3 grid_size(64,1,1);
  double *d_in, *d_out;
  hipMalloc((double**)&d_in, sizeof(double)*14*14*64);
  hipMalloc((double**)&d_out, sizeof(double)*7*7*64);
  hipMemcpy(d_in, in, sizeof(double)*14*14*64, hipMemcpyHostToDevice);

  pool_forward<<<grid_size, block_size>>>(d_in, d_out);

  hipMemcpy(out, d_out, sizeof(double)*7*7*64, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

__global__ void conv_forward(double* in, double* filter, double* bias, double* out) {
  int t_id = threadIdx.x + threadIdx.y*blockDim.x + blockDim.x*blockDim.y*blockIdx.x;
  int i_id = threadIdx.x+2 + threadIdx.y*(blockDim.x+4) + (blockDim.x+4)*(blockDim.y+4)*blockIdx.x;
  double res = 0;
  for (int i = -2; i <= 2; ++i) {
    for (int j = -2; j <= 2; ++j) {
      res += in[i_id+i*32+j]*filter[blockIdx.x*25+i*5+j];
    }
  }
  out[t_id] = res + bias[blockIdx.x];
  // printf("tid: %d\n", t_id);
}

void conv_forward_device_first(double* in, double* filter, double* bias, double* out) {
  double *d_i, *d_f, *d_b, *d_o;
  hipMalloc((double**)&d_i, sizeof(double)*32*32*1);
  hipMalloc((double**)&d_f, sizeof(double)*5*5*32);
  hipMalloc((double**)&d_b, sizeof(double)*32);
  hipMalloc((double**)&d_o, sizeof(double)*28*28*32);
  hipMemcpy(d_i, in, sizeof(double)*32*32*1, hipMemcpyHostToDevice);
  hipMemcpy(d_f, filter, sizeof(double)*5*5*32, hipMemcpyHostToDevice);
  hipMemcpy(d_b, bias, sizeof(double)*32, hipMemcpyHostToDevice);

  dim3 block_size(28,28,1);
  dim3 grid_size(32,1,1);
  conv_forward<<<grid_size, block_size>>>(d_i, d_f, d_b, d_o);

  hipMemcpy(out, d_o, sizeof(double)*28*28*32, hipMemcpyDeviceToHost);
  hipFree(d_i);
  hipFree(d_f);
  hipFree(d_b);
  hipFree(d_o);
}

__global__ void conv_forward_test(double* in, double* filter, double* bias, double* out) {
  int t_id = threadIdx.x + threadIdx.y*blockDim.x + blockDim.x*blockDim.y*blockIdx.x;
  int i_id = threadIdx.x+1 + threadIdx.y*(blockDim.x+2) + (blockDim.x+2)*(blockDim.y+2)*blockIdx.x;
  double res = 0;
  for (int i = -1; i <= 1; ++i) {
    for (int j = -1; j <= 1; ++j) {
      res += in[i_id+i*5+j]*filter[blockIdx.x*9+i*3+j];
    }
  }
  out[t_id] = res + bias[blockIdx.x];
  // printf("tid: %d\n", t_id);
}

void conv_forward_device_test(double* in, double* filter, double* bias, double* out) {
  double *d_i, *d_f, *d_b, *d_o;
  hipMalloc((double**)&d_i, sizeof(double)*5*5*1);
  hipMalloc((double**)&d_f, sizeof(double)*3*3*32);
  hipMalloc((double**)&d_b, sizeof(double)*32);
  hipMalloc((double**)&d_o, sizeof(double)*3*3*32);
  hipMemcpy(d_i, in, sizeof(double)*5*5*1, hipMemcpyHostToDevice);
  hipMemcpy(d_f, filter, sizeof(double)*3*3*32, hipMemcpyHostToDevice);
  hipMemcpy(d_b, bias, sizeof(double)*32, hipMemcpyHostToDevice);

  dim3 block_size(3,3,1);
  dim3 grid_size(32,1,1);
  conv_forward_test<<<grid_size, block_size>>>(d_i, d_f, d_b, d_o);

  hipMemcpy(out, d_o, sizeof(double)*3*3*32, hipMemcpyDeviceToHost);
  hipFree(d_i);
  hipFree(d_f);
  hipFree(d_b);
  hipFree(d_o);
}


// __global__ void add(int *x, int *y, int *z) {
//   z[threadIdx.x] = y[threadIdx.x] + x[threadIdx.x];
//   printf("Hello %d\n", threadIdx.x);
// }
//
// void test_device (int* x, int* y, int* z) {
//   int* d_x, *d_y, *d_z;
//   hipMalloc((int**)&d_x, sizeof(int)*16);
//   hipMalloc((int**)&d_y, sizeof(int)*16);
//   hipMalloc((int**)&d_z, sizeof(int)*16);
//   hipMemcpy(d_x, x, sizeof(int)*16, hipMemcpyHostToDevice);
//   hipMemcpy(d_y, y, sizeof(int)*16, hipMemcpyHostToDevice);
//
//   add<<<1,16>>> (d_x, d_y, d_z);
//
//   hipMemcpy(z, d_z, sizeof(int)*16, hipMemcpyDeviceToHost);
//   hipFree(d_x);
//   hipFree(d_y);
//   hipFree(d_z);
// }
