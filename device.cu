#include "hip/hip_runtime.h"
#include "main.h"

__global__ void full_forward(int* weight, int* input, int* output) {
  int index = threadIdx.x + blockDim.x*threadIdx.y;
  output[index] = weight[index] * input[index];
}

void full_device_forward(int * w, int * i, int * o) {
  printf("test of cuda forward function.");
  int *d_w, *d_i, *d_o;
  hipMalloc((int**)&d_w, sizeof(int)*16);
  hipMalloc((int**)&d_i, sizeof(int)*16);
  hipMalloc((int**)&d_o, sizeof(int)*16);
  hipMemcpy(d_w, w, sizeof(int)*16,hipMemcpyHostToDevice);
  hipMemcpy(d_i, i, sizeof(int)*16,hipMemcpyHostToDevice);
  hipMemcpy(d_o, o, sizeof(int)*16,hipMemcpyHostToDevice);
  dim3 block_size(1,1,1);
  dim3 thread_size(4,4,1);
  full_forward<<<block_size, thread_size>>>(d_w, d_i, d_o);
  hipMemcpy(w, d_w, sizeof(int)*16,hipMemcpyDeviceToHost);
  hipMemcpy(i, d_i, sizeof(int)*16,hipMemcpyDeviceToHost);
  hipMemcpy(o, d_o, sizeof(int)*16,hipMemcpyDeviceToHost);

  for (int e = 0; e < 16; ++e) {
    printf("%d\n", o[e]);
  }
}
// __global__ void conv_forward() {
//
// }
//
// __global__ void padding(float* origin, float* padded) {
//   // int blockId = blockIdx.x + blockDim.x*blockIdx.y;
//   // int threadId = threadIdx.x + threadDim.x*threadIdx.y + blockId*threadDim.x*threadDim.y;
//   int threadId = threadIdx.x + threadDim.x * threadidx.y;
//   if (threadIdx.x < 2 || threadIdx.x > 29 || threadIdx.y < 2 || threadIdx.y > 29) {
//     padded[threadId] = 0.0;
//     return;
//   }
//   int ori_id = (threadIdx.x-2) + (threadDim.x-2)*(threadIdx.y-2);
//   padded[threadId] = origin[ori_id];
// }
//
// template <typename IN_DIMS, size_t N_FILTERS>
// void
// conv_forward_device() {
//     // dim3 grids(2,2,1);
//     // dim3 blocks(16, 16, 1);
//     dim3 grids(1,1,1);
//     dim3 blocks(32, 32, 1);
//     float *in, *out;
//     float *d_in, *d_out;
//     in = (float*)malloc(sizeof(float)*28*28);
//     out = (float*)malloc(sizeof(float)*32*32);
//     hipMalloc((float**)&d_in, sizeof(float)*28*28);
//     hipMalloc((float**)&d_out, sizeof(float)*32*32);
//     hipMemcpy(d_in, in, sizeof(float)*28*28, hipMemcpyHostToDevice);
//     hipMemcpy(d_out, out, sizeof(float)*32*32, hipMemcpyHostToDevice);
//
//     padding<<<grids,blocks>>>(in, out);
//
//     __syncthreads();
//
//     hipMemcpy(d_out, out, sizeof(float)*32*32, hipMemcpyDeviceToHost);
//     free(in); free(out);
//     hipFree(d_in); hipFree(d_out);
//     // __syncthreads();
// }
