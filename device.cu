#include "hip/hip_runtime.h"
#include "main.h"

__global__ void pool_forward(double* in, double* out) {
  int t_id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*gridDim.x;
  int o_id = threadIdx.x/2 + threadIdx.y/2*(blockDim.x/2) + blockIdx.x*gridDim.x/4;
  if (out[o_id] < in[t_id]) {
    out[o_id] = in[t_id];
  }
}

void pool_device_forward(double* in, double* out) {
  dim3 block_size(8,8,1);
  dim3 grid_size(32,1,1);
  double *d_in, *d_out;
  hipMalloc((double**)&d_in, sizeof(double)*8*8*32);
  hipMalloc((double**)&d_out, sizeof(double)*4*4*32);
  hipMemcpy(d_in, in, sizeof(double)*8*8*32, hipMemcpyHostToDevice);

  pool_forward<<<grid_size, block_size>>>(d_in, d_out);

  hipMemcpy(out, d_out, sizeof(double)*4*4*32, hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

__global__ void add(int *x, int *y, int *z) {
  z[threadIdx.x] = y[threadIdx.x] + x[threadIdx.x];
  printf("Hello %d\n", threadIdx.x);
}

void test_device (int* x, int* y, int* z) {
  int* d_x, *d_y, *d_z;
  hipMalloc((int**)&d_x, sizeof(int)*16);
  hipMalloc((int**)&d_y, sizeof(int)*16);
  hipMalloc((int**)&d_z, sizeof(int)*16);
  hipMemcpy(d_x, x, sizeof(int)*16, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, sizeof(int)*16, hipMemcpyHostToDevice);

  add<<<1,16>>> (d_x, d_y, d_z);

  hipMemcpy(z, d_z, sizeof(int)*16, hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
}

// __global__ void conv_forward(double* weight, double* input, double* output) {
//
// }
//
// void conv_device_forward(double * w, double * i, double * o) {
//
// }



// __global__ void full_forward(double* weight, double* input, double* output) {
//   int threadId = threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;
//   int blockId = blockIdx.x;
//   output[blockId] += weight[threadId+blockId]*input[threadId];
// }
//
// void full_device_forward(double * w, double * i, double * o) {
//   printf("test of cuda forward function.\n");
//   double *d_w, *d_i, *d_o;
//   hipMalloc((double**)&d_w, sizeof(double)*7*7*64*1024);
//   hipMalloc((double**)&d_i, sizeof(double)*7*7*64);
//   hipMalloc((double**)&d_o, sizeof(double)*1024);
//   hipMemcpy(d_w, w, sizeof(double)*7*7*64*1024,hipMemcpyHostToDevice);
//   hipMemcpy(d_i, i, sizeof(double)*7*7*64,hipMemcpyHostToDevice);
//   hipMemcpy(d_o, o, sizeof(double)*1024,hipMemcpyHostToDevice);
//   dim3 grid_size(1024,1,1);
//   dim3 block_size(7,7,64);
//   full_forward<<<grid_size, block_size>>>(d_w, d_i, d_o);
//   hipMemcpy(o, d_o, sizeof(double)*1024,hipMemcpyDeviceToHost);
//   hipFree(d_w);
//   hipFree(d_i);
//   hipFree(d_o);
// }
// __global__ void conv_forward() {
//
// }
//
// __global__ void padding(double* origin, double* padded) {
//   // double blockId = blockIdx.x + blockDim.x*blockIdx.y;
//   // double threadId = threadIdx.x + threadDim.x*threadIdx.y + blockId*threadDim.x*threadDim.y;
//   double threadId = threadIdx.x + threadDim.x * threadidx.y;
//   if (threadIdx.x < 2 || threadIdx.x > 29 || threadIdx.y < 2 || threadIdx.y > 29) {
//     padded[threadId] = 0.0;
//     return;
//   }
//   double ori_id = (threadIdx.x-2) + (threadDim.x-2)*(threadIdx.y-2);
//   padded[threadId] = origin[ori_id];
// }
//
// template <typename IN_DIMS, size_t N_FILTERS>
// void
// conv_forward_device() {
//     // dim3 grids(2,2,1);
//     // dim3 blocks(7*7*64*1024, 7*7*64*1024, 1);
//     dim3 grids(1,1,1);
//     dim3 blocks(32, 32, 1);
//     double *in, *out;
//     double *d_in, *d_out;
//     in = (double*)malloc(sizeof(double)*8*8);
//     out = (double*)malloc(sizeof(double)*32*32);
//     hipMalloc((double**)&d_in, sizeof(double)*8*8);
//     hipMalloc((double**)&d_out, sizeof(double)*32*32);
//     hipMemcpy(d_in, in, sizeof(double)*8*8, hipMemcpyHostToDevice);
//     hipMemcpy(d_out, out, sizeof(double)*32*32, hipMemcpyHostToDevice);
//
//     padding<<<grids,blocks>>>(in, out);
//
//     __syncthreads();
//
//     hipMemcpy(d_out, out, sizeof(double)*32*32, hipMemcpyDeviceToHost);
//     free(in); free(out);
//     hipFree(d_in); hipFree(d_out);
//     // __syncthreads();
// }
