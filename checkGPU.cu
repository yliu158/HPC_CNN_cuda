#include "hip/hip_runtime.h"
#include <iostream>

int main()
{
    hipError_t cudaStatus;

    // 初获取设备数量
    int num = 0;
    cudaStatus = hipGetDeviceCount(&num);
    std::cout << "Number of GPU: " << num << std::endl;

    // 获取GPU设备属性
    hipDeviceProp_t prop;
    if (num > 0)
    {
        hipGetDeviceProperties(&prop, 0);
        // 打印设备名称
        std::cout << "Device: " <<prop.name << std::endl;
    }

    system("pause");
    return 0;
}
